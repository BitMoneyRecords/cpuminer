#include "hip/hip_runtime.h"
//
// Kernel that runs best on Kepler (Compute 3.0) devices
//
// NOTE: compile this .cu module for compute_11,sm_11 with --maxrregcount=124
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "spinlock_kernel.h"

#if WIN32
#ifdef _WIN64
#define _64BIT_ALIGN 1
#else
#define _64BIT_ALIGN 0
#endif
#else
#if __x86_64__
#define _64BIT_ALIGN 1
#else
#define _64BIT_ALIGN 0
#endif
#endif

// forward references
template <int WARPS_PER_BLOCK> __global__ void scrypt_core_kernel_spinlockA(uint32_t *g_idata, int *mutex);
template <int WARPS_PER_BLOCK> __global__ void scrypt_core_kernel_spinlockB(uint32_t *g_odata, int *mutex);
template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void scrypt_core_kernel_spinlockB_tex(uint32_t *g_odata, int *mutex);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

SpinlockKernel::SpinlockKernel() : KernelInterface()
{
}

bool SpinlockKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
    texRef1D_4_V.normalized = 0;
    texRef1D_4_V.filterMode = hipFilterModePoint;
    texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
    return true;
}

bool SpinlockKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
    texRef2D_4_V.normalized = 0;
    texRef2D_4_V.filterMode = hipFilterModePoint;
    texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
    texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
    return true;
}

bool SpinlockKernel::unbindtexture_1D()
{
    checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
    return true;
}

bool SpinlockKernel::unbindtexture_2D()
{
    checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
    return true;
}

void SpinlockKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool SpinlockKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, int *mutex, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    switch (WARPS_PER_BLOCK) {
        case 1: scrypt_core_kernel_spinlockA<1><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 2: scrypt_core_kernel_spinlockA<2><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 3: scrypt_core_kernel_spinlockA<3><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 4: scrypt_core_kernel_spinlockA<4><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 5: scrypt_core_kernel_spinlockA<5><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 6: scrypt_core_kernel_spinlockA<6><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 7: scrypt_core_kernel_spinlockA<7><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 8: scrypt_core_kernel_spinlockA<8><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        default: success = false; break;
    }

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    if (texture_cache)
    {
        if (texture_cache == 1)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: scrypt_core_kernel_spinlockB_tex<1,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 2: scrypt_core_kernel_spinlockB_tex<2,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 3: scrypt_core_kernel_spinlockB_tex<3,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 4: scrypt_core_kernel_spinlockB_tex<4,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 5: scrypt_core_kernel_spinlockB_tex<5,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 6: scrypt_core_kernel_spinlockB_tex<6,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 7: scrypt_core_kernel_spinlockB_tex<7,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 8: scrypt_core_kernel_spinlockB_tex<8,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                default: success = false; break;
            }
        }
        else if (texture_cache == 2)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: scrypt_core_kernel_spinlockB_tex<1,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 2: scrypt_core_kernel_spinlockB_tex<2,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 3: scrypt_core_kernel_spinlockB_tex<3,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 4: scrypt_core_kernel_spinlockB_tex<4,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 5: scrypt_core_kernel_spinlockB_tex<5,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 6: scrypt_core_kernel_spinlockB_tex<6,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 7: scrypt_core_kernel_spinlockB_tex<7,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 8: scrypt_core_kernel_spinlockB_tex<8,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                default: success = false; break;
            }
        } else success = false;
    }
    else
    {
        switch (WARPS_PER_BLOCK) {
            case 1: scrypt_core_kernel_spinlockB<1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 2: scrypt_core_kernel_spinlockB<2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 3: scrypt_core_kernel_spinlockB<3><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 4: scrypt_core_kernel_spinlockB<4><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 5: scrypt_core_kernel_spinlockB<5><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 6: scrypt_core_kernel_spinlockB<6><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 7: scrypt_core_kernel_spinlockB<7><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 8: scrypt_core_kernel_spinlockB<8><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            default: success = false; break;
        }
    }

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<7) | ((a00)>>25) );\
a1^=(((a10)<<7) | ((a10)>>25) );\
a2^=(((a20)<<7) | ((a20)>>25) );\
a3^=(((a30)<<7) | ((a30)>>25) );\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<9) | ((a00)>>23) );\
a1^=(((a10)<<9) | ((a10)>>23) );\
a2^=(((a20)<<9) | ((a20)>>23) );\
a3^=(((a30)<<9) | ((a30)>>23) );\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<13) | ((a00)>>19) );\
a1^=(((a10)<<13) | ((a10)>>19) );\
a2^=(((a20)<<13) | ((a20)>>19) );\
a3^=(((a30)<<13) | ((a30)>>19) );\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<18) | ((a00)>>14) );\
a1^=(((a10)<<18) | ((a10)>>14) );\
a2^=(((a20)<<18) | ((a20)>>14) );\
a3^=(((a30)<<18) | ((a30)>>14) );\
};\

static __device__ void xor_salsa8(uint32_t *B,uint32_t *C)
{
	uint32_t x[16];
	x[0]=(B[0] ^= C[0]);
	x[1]=(B[1] ^= C[1]);
	x[2]=(B[2] ^= C[2]);
	x[3]=(B[3] ^= C[3]);
	x[4]=(B[4] ^= C[4]);
	x[5]=(B[5] ^= C[5]);
	x[6]=(B[6] ^= C[6]);
	x[7]=(B[7] ^= C[7]);
	x[8]=(B[8] ^= C[8]);
	x[9]=(B[9] ^= C[9]);
	x[10]=(B[10] ^= C[10]);
	x[11]=(B[11] ^= C[11]);
	x[12]=(B[12] ^= C[12]);
	x[13]=(B[13] ^= C[13]);
	x[14]=(B[14] ^= C[14]);
	x[15]=(B[15] ^= C[15]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    B[ 0] += x[0]; B[ 1] += x[1]; B[ 2] += x[2]; B[ 3] += x[3]; B[ 4] += x[4]; B[ 5] += x[5]; B[ 6] += x[6]; B[ 7] += x[7];
    B[ 8] += x[8]; B[ 9] += x[9]; B[10] += x[10]; B[11] += x[11]; B[12] += x[12]; B[13] += x[13]; B[14] += x[14]; B[15] += x[15];
}

static __device__ uint4& operator^=(uint4& left, const uint4& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    left.z ^= right.z;
    left.w ^= right.w;
    return left;
}

static __device__ void lock(int *mutex, int i)
{
    while( atomicCAS( &mutex[i], 0, 1 ) != 0 )
    {
        // keep the (slow) special function unit busy to avoid hammering
        // the memory controller with atomic operations while busy waiting
        asm volatile("{\t\n.reg .f32 tmp;\t\n"
                     "lg2.approx.f32 tmp, 0f00000000;\t\n}" :: );
    }
}

static __device__ void unlock(int *mutex, int i)
{
    atomicExch( &mutex[i], 0 );
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel using spinlocks to cut shared memory use in half.
//! Ideal for Kepler devices where shared memory use prevented optimal occupancy.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernel_spinlockA(uint32_t *g_idata, int *mutex)
{
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx        = threadIdx.x / warpSize;
    volatile int warpThread     = threadIdx.x % warpSize;

    // add block specific offsets
    volatile int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32 * offset;
    uint32_t * volatile V = c_V[offset / WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/4;
    volatile unsigned int Z = 4*(warpThread%4);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&V[SCRATCH*(wu+Y)+Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z])) = *((uint4*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx/2][warpThread][idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&V[SCRATCH*(wu+Y)+16+Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z])) = *((uint4*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx/2][warpThread][idx];

    for (int i = 1; i < 1024; i++) {

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&V[SCRATCH*(wu+Y) + i*32 + Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z]));

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&V[SCRATCH*(wu+Y) + i*32 + 16 + Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z]));
    }
    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
}

template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernel_spinlockB(uint32_t *g_odata, int *mutex)
{
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx        = threadIdx.x / warpSize;
    volatile int warpThread     = threadIdx.x % warpSize;

    // add block specific offsets
    volatile int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;
    uint32_t * volatile V = c_V[offset / WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/4;
    volatile unsigned int Z = 4*(warpThread%4);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&X[warpIdx/2][wu+Y][Z])) = *((uint4*)(&V[SCRATCH*(wu+Y) + 1023*32 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx/2][warpThread][idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&X[warpIdx/2][wu+Y][Z])) = *((uint4*)(&V[SCRATCH*(wu+Y) + 1023*32 + 16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx/2][warpThread][idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);

    for (int i = 0; i < 1024; i++) {

        X[warpIdx/2][warpThread][16] = C[0];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&X[warpIdx/2][wu+Y][Z])) ^= *((uint4*)(&V[SCRATCH*(wu+Y) + 32*(X[warpIdx/2][wu+Y][16] & 1023) + Z]));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx/2][warpThread][idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&X[warpIdx/2][wu+Y][Z])) ^= *((uint4*)(&V[SCRATCH*(wu+Y) + 32*(X[warpIdx/2][wu+Y][16] & 1023) + 16 + Z]));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx/2][warpThread][idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = B[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&g_odata[32*(wu+Y)+Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z]));

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = C[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&g_odata[32*(wu+Y)+16+Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z]));

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
}

template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void
scrypt_core_kernel_spinlockB_tex(uint32_t *g_odata, int *mutex)
{
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx        = threadIdx.x / warpSize;
    volatile int warpThread     = threadIdx.x % warpSize;

    // add block specific offsets
    volatile int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/4;
    volatile unsigned int Z = 4*(warpThread%4);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&X[warpIdx/2][wu+Y][Z])) = ((TEX_DIM == 1) ?
                    tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + 1023*32 + Z)/4) :
                    tex2D(texRef2D_4_V, 0.5f + (32*1023 + Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx/2][warpThread][idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&X[warpIdx/2][wu+Y][Z])) = ((TEX_DIM == 1) ?
                    tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + 1023*32 + 16+Z)/4) :
                    tex2D(texRef2D_4_V, 0.5f + (32*1023 + 16+Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx/2][warpThread][idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);

    for (int i = 0; i < 1024; i++) {

        X[warpIdx/2][warpThread][16] = C[0];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&X[warpIdx/2][wu+Y][Z])) ^= ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + 32*(X[warpIdx/2][wu+Y][16] & 1023) + Z)/4) :
                        tex2D(texRef2D_4_V, 0.5f + (32*(X[warpIdx/2][wu+Y][16] & 1023) + Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx/2][warpThread][idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&X[warpIdx/2][wu+Y][Z])) ^= ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + 32*(X[warpIdx/2][wu+Y][16] & 1023) + 16+Z)/4) :
                        tex2D(texRef2D_4_V, 0.5f + (32*(X[warpIdx/2][wu+Y][16] & 1023) + 16+Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx/2][warpThread][idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = B[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&g_odata[32*(wu+Y)+Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z]));

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx/2][warpThread][idx] = C[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&g_odata[32*(wu+Y)+16+Z])) = *((uint4*)(&X[warpIdx/2][wu+Y][Z]));

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx/2);
}
