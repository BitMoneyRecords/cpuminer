#include "hip/hip_runtime.h"
//
// Kernel that runs best on Kepler (Compute 3.5) devices
//
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=64
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "titan_kernel.h"

// forward references
__global__ void scrypt_core_kernel_spinlock_titanA(uint32_t *g_idata, int *mutex);
__global__ void scrypt_core_kernel_spinlock_titanB(uint32_t *g_odata, int *mutex);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

TitanKernel::TitanKernel() : KernelInterface()
{
}

bool TitanKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
    return true;
}

bool TitanKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
    return true;
}

bool TitanKernel::unbindtexture_1D()
{
    return true;
}

bool TitanKernel::unbindtexture_2D()
{
    return true;
}

void TitanKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool TitanKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, int *mutex, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    scrypt_core_kernel_spinlock_titanA<<< grid, threads, 0, stream >>>(d_idata, mutex);

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    scrypt_core_kernel_spinlock_titanB<<< grid, threads, 0, stream >>>(d_odata, mutex);

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

#define ROTL(a, b) __funnelshift_l( a, a, b );

static __device__ __forceinline__ void lock(int *mutex, int i)
{
    while( atomicCAS( &mutex[i], 0, 1 ) != 0 );
}

static __device__ __forceinline__ void unlock(int *mutex, int i)
{
    atomicExch( &mutex[i], 0 );
}

static __device__ __forceinline__ void xor_salsa8(uint32_t *B, const uint32_t *C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

static __device__ __forceinline__ uint2& operator^=(uint2& left, const uint2& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    return left;
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel with spinlock guards around a smaller shared memory
//! Version for Geforce Titan, low register count (<=64), low shared mem use.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
scrypt_core_kernel_spinlock_titanA(uint32_t *g_idata, int *mutex)
{
    volatile __shared__ uint32_t X[WU_PER_WARP][16+2]; // +2 to reduce bank conflicts
                                                       // while maintaining alignment
    int warpIdx         = threadIdx.x / warpSize;
    int warpThread      = threadIdx.x % warpSize;
    int WARPS_PER_BLOCK = blockDim.x / warpSize;

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32 * offset;
    uint32_t* V = (uint32_t*)c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/8;
    volatile unsigned int Z = 2*(warpThread%8);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x);
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&V[SCRATCH*(wu+Y)+Z])) = *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpThread][idx];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&V[SCRATCH*(wu+Y)+16+Z])) = *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpThread][idx];

    for (int i = 1; i < 1024; i++) {

        if (warpThread == 0) unlock(mutex, blockIdx.x);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + Z])) = *((uint2*)(&X[wu+Y][Z]));

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + 16 + Z])) = *((uint2*)(&X[wu+Y][Z]));
    }
    if (warpThread == 0) unlock(mutex, blockIdx.x);
}

__global__ void
scrypt_core_kernel_spinlock_titanB(uint32_t *g_odata, int *mutex)
{
    volatile __shared__ uint32_t X[WU_PER_WARP][16+2]; // +2 to reduce bank conflicts
                                                       // while maintaining alignment
    int warpIdx         = threadIdx.x / warpSize;
    int warpThread      = threadIdx.x % warpSize;
    int WARPS_PER_BLOCK = blockDim.x / warpSize;

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;
    const uint32_t* __restrict__ V = (const uint32_t*)c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/8;
    volatile unsigned int Z = 2*(warpThread%8);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x);
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpThread][idx];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + 16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpThread][idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x);

    for (int i = 0; i < 1024; i++) {

        X[warpThread][16] = C[0];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&X[wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[wu+Y][16] & 1023) + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpThread][idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&X[wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[wu+Y][16] & 1023) + 16 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpThread][idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)(&X[wu+Y][Z]));

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&g_odata[32*(wu+Y)+16+Z])) = *((uint2*)(&X[wu+Y][Z]));

    if (warpThread == 0) unlock(mutex, blockIdx.x);
}
