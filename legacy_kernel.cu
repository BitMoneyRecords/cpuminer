#include "hip/hip_runtime.h"
//
// Kernel that runs best on Legacy (Compute 1.x) devices
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=124
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "legacy_kernel.h"

#if WIN32
#ifdef _WIN64
#define _64BIT_ALIGN 1
#else
#define _64BIT_ALIGN 0
#endif
#else
#if __x86_64__
#define _64BIT_ALIGN 1
#else
#define _64BIT_ALIGN 0
#endif
#endif

// forward references
template <int WARPS_PER_BLOCK> __global__ void scrypt_core_kernelA(uint32_t *g_idata);
template <int WARPS_PER_BLOCK> __global__ void scrypt_core_kernelB(uint32_t *g_odata);
template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void scrypt_core_kernelB_tex(uint32_t *g_odata);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

// using texture references for the "tex" variants of the B kernels
texture<uint2, 1, hipReadModeElementType> texRef1D_2_V;
texture<uint2, 2, hipReadModeElementType> texRef2D_2_V;

LegacyKernel::LegacyKernel() : KernelInterface()
{
}

bool LegacyKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<uint2>();
    texRef1D_2_V.normalized = 0;
    texRef1D_2_V.filterMode = hipFilterModePoint;
    texRef1D_2_V.addressMode[0] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture(NULL, &texRef1D_2_V, d_V, &channelDesc2, size));
    return true;
}

bool LegacyKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<uint2>();
    texRef2D_2_V.normalized = 0;
    texRef2D_2_V.filterMode = hipFilterModePoint;
    texRef2D_2_V.addressMode[0] = hipAddressModeClamp;
    texRef2D_2_V.addressMode[1] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_2_V, d_V, &channelDesc2, width, height, pitch));
    return true;
}

bool LegacyKernel::unbindtexture_1D()
{
    checkCudaErrors(hipUnbindTexture(texRef1D_2_V));
    return true;
}

bool LegacyKernel::unbindtexture_2D()
{
    checkCudaErrors(hipUnbindTexture(texRef2D_2_V));
    return true;
}

void LegacyKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool LegacyKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, int *mutex, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    switch (WARPS_PER_BLOCK) {
        case 1: scrypt_core_kernelA<1><<< grid, threads, 0, stream >>>(d_idata); break;
        case 2: scrypt_core_kernelA<2><<< grid, threads, 0, stream >>>(d_idata); break;
        case 3: scrypt_core_kernelA<3><<< grid, threads, 0, stream >>>(d_idata); break;
//            case 4: scrypt_core_kernelA<4><<< grid, threads, 0, stream >>>(d_idata); break;
//            case 5: scrypt_core_kernelA<5><<< grid, threads, 0, stream >>>(d_idata); break;
//            case 6: scrypt_core_kernelA<6><<< grid, threads, 0, stream >>>(d_idata); break;
//            case 7: scrypt_core_kernelA<7><<< grid, threads, 0, stream >>>(d_idata); break;
//            case 8: scrypt_core_kernelA<8><<< grid, threads, 0, stream >>>(d_idata); break;
        default: success = false; break;
    }

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    if (texture_cache)
    {
        if (texture_cache == 1)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: scrypt_core_kernelB_tex<1,1><<< grid, threads, 0, stream >>>(d_odata); break;
                case 2: scrypt_core_kernelB_tex<2,1><<< grid, threads, 0, stream >>>(d_odata); break;
                case 3: scrypt_core_kernelB_tex<3,1><<< grid, threads, 0, stream >>>(d_odata); break;
//                    case 4: scrypt_core_kernelB_tex<4,1><<< grid, threads, 0, stream >>>(d_odata); break;
//                    case 5: scrypt_core_kernelB_tex<5,1><<< grid, threads, 0, stream >>>(d_odata); break;
//                    case 6: scrypt_core_kernelB_tex<6,1><<< grid, threads, 0, stream >>>(d_odata); break;
//                    case 7: scrypt_core_kernelB_tex<7,1><<< grid, threads, 0, stream >>>(d_odata); break;
//                    case 8: scrypt_core_kernelB_tex<8,1><<< grid, threads, 0, stream >>>(d_odata); break;
                default: success = false; break;
            }
        }
        else if (texture_cache == 2)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: scrypt_core_kernelB_tex<1,2><<< grid, threads, 0, stream >>>(d_odata); break;
                case 2: scrypt_core_kernelB_tex<2,2><<< grid, threads, 0, stream >>>(d_odata); break;
                case 3: scrypt_core_kernelB_tex<3,2><<< grid, threads, 0, stream >>>(d_odata); break;
//                   case 4: scrypt_core_kernelB_tex<4,2><<< grid, threads, 0, stream >>>(d_odata); break;
//                   case 5: scrypt_core_kernelB_tex<5,2><<< grid, threads, 0, stream >>>(d_odata); break;
//                   case 6: scrypt_core_kernelB_tex<6,2><<< grid, threads, 0, stream >>>(d_odata); break;
//                   case 7: scrypt_core_kernelB_tex<7,2><<< grid, threads, 0, stream >>>(d_odata); break;
//                   case 8: scrypt_core_kernelB_tex<8,2><<< grid, threads, 0, stream >>>(d_odata); break;
                default: success = false; break;
            }
        } else success = false;
    }
    else
    {
        switch (WARPS_PER_BLOCK) {
            case 1: scrypt_core_kernelB<1><<< grid, threads, 0, stream >>>(d_odata); break;
            case 2: scrypt_core_kernelB<2><<< grid, threads, 0, stream >>>(d_odata); break;
            case 3: scrypt_core_kernelB<3><<< grid, threads, 0, stream >>>(d_odata); break;
//                case 4: scrypt_core_kernelB<4><<< grid, threads, 0, stream >>>(d_odata); break;
//                case 5: scrypt_core_kernelB<5><<< grid, threads, 0, stream >>>(d_odata); break;
//                case 6: scrypt_core_kernelB<6><<< grid, threads, 0, stream >>>(d_odata); break;
//                case 7: scrypt_core_kernelB<7><<< grid, threads, 0, stream >>>(d_odata); break;
//                case 8: scrypt_core_kernelB<8><<< grid, threads, 0, stream >>>(d_odata); break;
            default: success = false; break;
        }
    }

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

static __host__ __device__ void xor_salsa8(uint32_t * const B, const uint32_t * const C)
{
    // the "volatile" puts data into registers right away
    volatile uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    volatile uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    volatile uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    volatile uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    for (int i = 0; i < 4; ++i)
    {
        /* Operate on columns. */
        x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
        x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
        xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
        x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
        /* Operate on rows. */
        x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
        x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
        x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
        x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);
    }

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

static __host__ __device__ uint2& operator^=(uint2& left, const uint2& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    return left;
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel with higher shared memory use (faster on older devices)
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernelA(uint32_t *g_idata)
{
    __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][32+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx    = threadIdx.x / warpSize;
    volatile int warpThread = threadIdx.x % warpSize;

    // add block specific offsets
    volatile int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32      * offset;
    uint32_t * volatile V = c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/16;
    volatile unsigned int Z = 2*(warpThread%16);

    {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+Z]));

#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&V[SCRATCH*(wu+Y) + 0*32 + Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));

        for (int i = 1; i < 1024; i++)
        {
            xor_salsa8(&X[warpIdx][warpThread][0], &X[warpIdx][warpThread][16]);
            xor_salsa8(&X[warpIdx][warpThread][16], &X[warpIdx][warpThread][0]);

#pragma unroll 16
            for (int wu=0; wu < 32; wu+=2)
                *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));
        }
    }
}

template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernelB(uint32_t *g_odata)
{
    __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][32+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx    = threadIdx.x / warpSize;
    volatile int warpThread = threadIdx.x % warpSize;

    // add block specific offsets
    volatile int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32      * offset;
    uint32_t * volatile V = c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/16;
    volatile unsigned int Z = 2*(warpThread%16);

    {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + Z]));

        xor_salsa8(&X[warpIdx][warpThread][0], &X[warpIdx][warpThread][16]);
        xor_salsa8(&X[warpIdx][warpThread][16], &X[warpIdx][warpThread][0]);

        for (int i = 0; i < 1024; i++)
        {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[warpIdx][wu+Y][16] & 1023) + Z]));

            xor_salsa8(&X[warpIdx][warpThread][0], &X[warpIdx][warpThread][16]);
            xor_salsa8(&X[warpIdx][warpThread][16], &X[warpIdx][warpThread][0]);
        }

#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));
    }
}

template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void
scrypt_core_kernelB_tex(uint32_t *g_odata)
{
    __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][32+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx    = threadIdx.x / warpSize;
    volatile int warpThread = threadIdx.x % warpSize;

    // add block specific offsets
    volatile int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32      * offset;
    uint32_t * volatile V = c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/16;
    volatile unsigned int Z = 2*(warpThread%16);

    {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) = ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_2_V, (SCRATCH*(offset+wu+Y) + 1023*32 + Z)/2) :
                        tex2D(texRef2D_2_V, 0.5f + (32*1023 + Z)/2, 0.5f + (offset+wu+Y)));

        xor_salsa8(&X[warpIdx][warpThread][0], &X[warpIdx][warpThread][16]);
        xor_salsa8(&X[warpIdx][warpThread][16], &X[warpIdx][warpThread][0]);

        for (int i = 0; i < 1024; i++)
        {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) ^= ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_2_V, (SCRATCH*(offset+wu+Y) + 32*(X[warpIdx][wu+Y][16] & 1023) + Z)/2) :
                        tex2D(texRef2D_2_V, 0.5f + (32*(X[warpIdx][wu+Y][16] & 1023) + Z)/2, 0.5f + (offset+wu+Y)));

            xor_salsa8(&X[warpIdx][warpThread][0], &X[warpIdx][warpThread][16]);
            xor_salsa8(&X[warpIdx][warpThread][16], &X[warpIdx][warpThread][0]);
        }

#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));
    }
}

